#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream> 
#include <string> 
using namespace std;
#include<cstdlib>
#include <hip/hip_runtime.h>
#include<string>
#include <hip/hip_runtime.h>
#include <math.h>
#include <SDL.h>
#include <SDL_image.h>
#undef main



__global__ void next_species(int* galive, int* ggalive, int* gage,int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int count = 0;
    if (galive[(id - 1)%n] % 2 != 0)   count += 1; //left
    if (galive[(id + 1) % n] % 2 != 0)   count += 1; //right
    if (galive[(id + blockDim.x) % n] % 2 != 0)   count += 1; //down
    if (galive[(id - blockDim.x) % n] % 2 != 0)   count += 1; //up
    if (galive[(id + blockDim.x-1) % n] % 2 != 0)   count += 1; //down left
    if (galive[(id - blockDim.x-1) % n] % 2 != 0)   count += 1; //up left
    if (galive[(id + blockDim.x+1) % n] % 2 != 0)   count += 1; //down right
    if (galive[(id - blockDim.x+1) % n] % 2 != 0)   count += 1; //up right
    //printf("%d--%d--%d\n", id, count, galive[id]);

    if (galive[id] % 2 != 0) {
        if (count != 2 && count != 3) {
            ggalive[id] = 0;
        }
        else {
            if (gage[id * 3] != 255 || gage[id * 3 + 1] != 0 || gage[id * 3 + 2] != 0) {
                
                if (gage[id * 3] != 0 && gage[id * 3 + 1] == 255 && gage[id * 3 + 2] == 0)   gage[id * 3] -= 51;
                else if (gage[id * 3] == 0 && gage[id * 3 + 1] == 255 && gage[id * 3 + 2] != 255)   gage[id * 3+2] += 51;
                else if (gage[id * 3] == 0 && gage[id * 3 + 1] != 0 && gage[id * 3 + 2] == 255)   gage[id * 3+1] -= 51;
                else if (gage[id * 3] != 255 && gage[id * 3 + 1] == 0 && gage[id * 3 + 2] == 255)   gage[id * 3 ] += 51;
                else if (gage[id * 3] == 255 && gage[id * 3 + 1] == 0 && gage[id * 3 + 2] != 0)   gage[id * 3+2] -= 51;
            }
        }
    }
    else {
        if (count == 3) {
            ggalive[id] = 1;
            gage[id * 3] = 255;
            gage[id * 3+1] = 255;
            gage[id * 3+2] = 0;


        }
    }

};




//Screen dimension constants
int SCREEN_WIDTH = 720;
int SCREEN_HEIGHT = 480;



bool init();

//Loads media
bool loadMedia();

//Frees media and shuts down SDL
void close();

//Loads individual image as texture
SDL_Texture* loadTexture(std::string path);

//The window we'll be rendering to
SDL_Window* gWindow = NULL;

//The window renderer
SDL_Renderer* gRenderer = NULL;

bool init()
{
    //Initialization flag
    bool success = true;

    //Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0)
    {
        printf("SDL could not initialize! SDL Error: %s\n", SDL_GetError());
        success = false;
    }
    else
    {
        //Set texture filtering to linear
        if (!SDL_SetHint(SDL_HINT_RENDER_SCALE_QUALITY, "1"))
        {
            printf("Warning: Linear texture filtering not enabled!");
        }

        //Create window
        gWindow = SDL_CreateWindow("SDL Tutorial", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
        if (gWindow == NULL)
        {
            printf("Window could not be created! SDL Error: %s\n", SDL_GetError());
            success = false;
        }
        else
        {
            //Create renderer for window
            gRenderer = SDL_CreateRenderer(gWindow, -1, SDL_RENDERER_ACCELERATED);
            if (gRenderer == NULL)
            {
                printf("Renderer could not be created! SDL Error: %s\n", SDL_GetError());
                success = false;
            }
            else
            {
                //Initialize renderer color
                SDL_SetRenderDrawColor(gRenderer, 0xFF, 0xFF, 0xFF, 0xFF);

                //Initialize PNG loading
                int imgFlags = IMG_INIT_PNG;
                if (!(IMG_Init(imgFlags) & imgFlags))
                {
                    printf("SDL_image could not initialize! SDL_image Error: %s\n", IMG_GetError());
                    success = false;
                }
            }
        }
    }

    return success;
}

bool loadMedia()
{
    //Loading success flag
    bool success = true;

    //Nothing to load
    return success;
}

void close()
{
    //Destroy window	
    SDL_DestroyRenderer(gRenderer);
    SDL_DestroyWindow(gWindow);
    gWindow = NULL;
    gRenderer = NULL;

    //Quit SDL subsystems
    IMG_Quit();
    SDL_Quit();
}

SDL_Texture* loadTexture(std::string path)
{
    //The final texture
    SDL_Texture* newTexture = NULL;

    //Load image at specified path
    SDL_Surface* loadedSurface = IMG_Load(path.c_str());
    if (loadedSurface == NULL)
    {
        printf("Unable to load image %s! SDL_image Error: %s\n", path.c_str(), IMG_GetError());
    }
    else
    {
        //Create texture from surface pixels
        newTexture = SDL_CreateTextureFromSurface(gRenderer, loadedSurface);
        if (newTexture == NULL)
        {
            printf("Unable to create texture from %s! SDL Error: %s\n", path.c_str(), SDL_GetError());
        }

        //Get rid of old loaded surface
        SDL_FreeSurface(loadedSurface);
    }

    return newTexture;
}


int main() {
    int cell_size = 10,temp;
    int alive[720/10][480/10];
    int age[720 / 10][480 / 10][3];
    FILE* fp;
    fp = fopen("somefile.txt", "r");

    for (int i = 0; i < SCREEN_WIDTH / cell_size; i++) {
        for (int j = 0; j < SCREEN_HEIGHT / cell_size; j++) {
            temp = rand();
            char str[1];
            if(temp%2==0)   alive[i][j] = 0;
            else   alive[i][j] = 1;
            //fscanf(fp, "%s", str);
            //alive[i][j]=stoi(str);
           
            age[i][j][0] = 255;
            age[i][j][1] = 255;
            age[i][j][2] = 0;
        }
    }
    fclose(fp);
    int* galive, * gage, *ggalive;
    hipMalloc(&galive, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size) * sizeof(int));
    hipMalloc(&ggalive, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size) * sizeof(int));
    hipMalloc(&gage, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size)*3 * sizeof(int));

    //Start up SDL and create window
	if( !init() )
	{
		printf( "Failed to initialize!\n" );
	}
	else
	{
		//Load media
		if( !loadMedia() )
		{
			printf( "Failed to load media!\n" );
		}
		else
		{	
			//Main loop flag
			bool quit = false;

			//Event handler
			SDL_Event e;

			//While application is running
			while( !quit )
			{
                /*for (int i = 0; i < SCREEN_WIDTH / cell_size; i++) {
                    printf("\n");
                    for (int j = 0; j < SCREEN_HEIGHT / cell_size; j++) {
                        printf("%d ", alive[i][j]);
                    }
                }printf("\n");*/
				//Handle events on queue
				while( SDL_PollEvent( &e ) != 0 )
				{
					//User requests quit
					if( e.type == SDL_QUIT )
					{
						quit = true;
					}
				}

				//Clear screen
				SDL_SetRenderDrawColor( gRenderer, 0xFF, 0xFF, 0xFF, 0xFF );
				SDL_RenderClear( gRenderer );

				//Render red filled quad
				//SDL_Rect fillRect = { SCREEN_WIDTH / 4, SCREEN_HEIGHT / 4, SCREEN_WIDTH / 2, SCREEN_HEIGHT / 2 };
				//SDL_SetRenderDrawColor( gRenderer, 0xFF, 0x00, 0x00, 0xFF );		
				//SDL_RenderFillRect( gRenderer, &fillRect );
                for (int i = 0; i < SCREEN_WIDTH/cell_size; i++) {
                    for (int j = 0; j < SCREEN_HEIGHT/cell_size; j++) {
                        if (alive[i][j] % 2 == 0) {
                            continue;
                        }
                        SDL_Rect fillRect = { i* cell_size, j * cell_size,cell_size, cell_size };
                        SDL_SetRenderDrawColor(gRenderer, age[i][j][0], age[i][j][1], age[i][j][2], 0xFF);
                        SDL_RenderFillRect(gRenderer, &fillRect); 
                    }
                }

				//Render green outlined quad
				/*SDL_Rect outlineRect = { SCREEN_WIDTH / 6, SCREEN_HEIGHT / 6, SCREEN_WIDTH * 2 / 3, SCREEN_HEIGHT * 2 / 3 };
				SDL_SetRenderDrawColor( gRenderer, 0x00, 0xFF, 0x00, 0xFF );		
				SDL_RenderDrawRect( gRenderer, &outlineRect );*/
				
				//Draw blue horizontal line
				SDL_SetRenderDrawColor( gRenderer, 0x00, 0, 255, 0xFF );		
				//SDL_RenderDrawLine( gRenderer, 0, SCREEN_HEIGHT / 2, SCREEN_WIDTH, SCREEN_HEIGHT / 2 );
                for (int i = 0; i < SCREEN_WIDTH/cell_size; i++) {
                    SDL_RenderDrawLine(gRenderer, i*cell_size, 0 , i*cell_size, SCREEN_HEIGHT );
                }
                for (int i = 0; i < SCREEN_HEIGHT/cell_size; i++) {
                    SDL_RenderDrawLine(gRenderer, 0, i*cell_size,SCREEN_WIDTH, i * cell_size);
                }
				//Draw vertical line of yellow dots
				SDL_SetRenderDrawColor( gRenderer, 0xFF, 0xFF, 0x00, 0xFF );
				/*for( int i = 0; i < SCREEN_HEIGHT; i += 4 )
				{
					SDL_RenderDrawPoint( gRenderer, SCREEN_WIDTH / 2, i );
				}*/

				//Update screen
				SDL_RenderPresent( gRenderer );
                hipMemcpy(galive, alive, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size) * sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(ggalive, alive, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size) * sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(gage, age, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size)*3 * sizeof(int), hipMemcpyHostToDevice);
                next_species << <  SCREEN_WIDTH / cell_size, SCREEN_HEIGHT / cell_size  >> > (galive, ggalive, gage, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size));
                hipMemcpy(alive, ggalive, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size) * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(age, gage, (SCREEN_WIDTH / cell_size) * (SCREEN_HEIGHT / cell_size)*3 * sizeof(int), hipMemcpyDeviceToHost);
                SDL_Delay(50);
  
			}
		}
	}

	//Free resources and close SDL
	close();

    return 0;
}